#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifndef __HIPCC__
#include ""
#include "hip/hip_runtime_api.h"
#endif

#ifdef __INTELLISENSE__
void __syncthreads() {}
#endif

#include "kernel.h"
#include "properties.h"

#include <iostream>
using namespace std;

static double* pos[3];
static double* vel[3];
static double* acc[3];
static double* energy;

struct vec {
	double* v[3];
	__device__ void set(int i, double3 p) {
		v[X][i] = p.x;
		v[Y][i] = p.y;
		v[Z][i] = p.z;
	}
	__device__ double3 get(int i) {
		return double3({ v[X][i],v[Y][i],v[Z][i] });
	}
	vec(double* p[3]) {
		for (int i = 0; i < 3; i++)
			v[i] = p[i];
	}
};

void gpu_alloc() {
	for (int i = 0; i < 3; i++) {
		hipMalloc(&pos[i], AMOUNT * sizeof(double));
		hipMalloc(&vel[i], AMOUNT * sizeof(double));
		hipMalloc(&acc[i], AMOUNT * sizeof(double));
		hipMemset(acc[i], 0, AMOUNT * sizeof(double));
	}
	hipMalloc(&energy, AMOUNT * sizeof(double));
}
void gpu_dealloc() {
	for (int i = 0; i < 3; i++) {
		hipFree(pos[i]);
		hipFree(vel[i]);
		hipFree(acc[i]);
	}
	hipFree(energy);
	hipDeviceReset();
}

bool pos_valid = false, vel_valid = false, energy_valid = false;

void get_pos() {
	if (pos_valid)
		return;
	for(int i = 0; i < 3; i++)
		hipMemcpy(_pos[i], pos[i], AMOUNT * sizeof(double), hipMemcpyDeviceToHost);
	pos_valid = true;
}
void get_vel() {
	if (vel_valid)
		return;
	for (int i = 0; i < 3; i++)
		hipMemcpy(_vel[i], vel[i], AMOUNT * sizeof(double), hipMemcpyDeviceToHost);
	vel_valid = true;
}

void set_pos() {
	for (int i = 0; i < 3; i++)
		hipMemcpy(pos[i], _pos[i], AMOUNT * sizeof(double), hipMemcpyHostToDevice);
	pos_valid = true;
}
void set_vel() {
	for (int i = 0; i < 3; i++)
		hipMemcpy(vel[i], _vel[i], AMOUNT * sizeof(double), hipMemcpyHostToDevice);
	vel_valid = true;
}

void print_err() {
	hipDeviceSynchronize();
	cout << hipGetErrorString(hipGetLastError()) << endl;
}

__device__ double hypot2(double3 p) {
	return p.x * p.x + p.y * p.y + p.z * p.z;
}
__device__ double3 round(double3 a) {
	return { round(a.x),round(a.y),round(a.z) };
}

#ifndef __HIPCC__
__device__ double3& operator-= (double3& a, double3 b) {
	a.x -= b.x;
	a.y -= b.y;
	a.z -= b.z;
	return a;
}
__device__ double3& operator+=(double3& a, double3 b) {
	a.x += b.x;
	a.y += b.y;
	a.z += b.z;
	return a;
}
#endif

__device__ double3& operator*= (double3& a, double b) {
	a.x *= b;
	a.y *= b;
	a.z *= b;
	return a;
}
__device__ double3 operator- (double3 a, double3 b) {
	return { a.x - b.x, a.y - b.y, a.z - b.z };
}
__device__ double3 operator* (double b, double3 a) {
	return { a.x * b, a.y * b, a.z * b };
}
__device__ double3 operator+ (double3 a, double3 b) {
	return { a.x + b.x, a.y + b.y, a.z + b.z };
}

__device__ bool operator== (double3 a, double3 b) {
	return a.x == b.x && a.y == b.y && a.z == b.z;
}

#define GPU_PAIR_INTERACTION_WRAPPER( __CODE__ ) 				\
																\
	int tid = threadIdx.x,										\
	bid = blockIdx.x,											\
	ind = bid * BLOCK_SIZE + tid;								\
																\
	double3 p = 1. / SIZE * pos.get(ind),						\
	v = vel.get(ind);											\
																\
	__shared__ double3 _pos[BLOCK_SIZE];						\
	for (int i = 0; i < GRID_SIZE; i++) {						\
																\
		__syncthreads();										\
		_pos[tid] = 1. / SIZE * pos.get(i * BLOCK_SIZE + tid);	\
		__syncthreads();										\
																\
		for (int j = 0; j < BLOCK_SIZE; j++) {					\
			double3 _p = _pos[j];								\
			if (i != bid || j != tid) {							\
				__CODE__										\
			}													\
		}														\
	}															\
																\
	p *= SIZE;

constexpr double ss_ss = (SIZE * SIZE) / (SIGMA * SIGMA);

__device__ void get_a(double3& a_lj, double3& a_em, double3 p, double3 _p) {
	double3 d = p - _p;
	d -= round(d);

	double d2 = hypot2(d);

#ifdef ENABLE_LJ
	double r2 = d2 * ss_ss,
		r_2 = 1. / r2,
		r_4 = r_2 * r_2,
		r_6 = r_4 * r_2,
		r_8 = r_4 * r_4,
		_2r_14__r_8 = (r_6 - .5) * r_8;
	a_lj += (_2r_14__r_8 * d);
#endif
	
#ifdef ENABLE_EM
	double d_2 = 1 / d2,
		d_1 = sqrt(d_2);
	a_em += d_2 * d_1 * d;
#endif 
}
__device__ void get_e(double& e_lj, double& e_em, double3 p, double3 _p) {
	double3 d = p - _p;
	d -= round(d);
	double d2 = hypot2(d);

#ifdef ENABLE_LJ
	double r2 = d2 * ss_ss,
		r_2 = 1. / r2,
		r_4 = r_2 * r_2,
		r_6 = r_4 * r_2;
	e_lj += (r_6 - 1) * r_6;
#endif

#ifdef ENABLE_EM
	double d_1 = 1 / sqrt(d2);
	e_em += d_1;
#endif
}

__global__ void euler_gpu(vec pos, vec vel, vec acc) {
	double3 a_lj = { 0., 0., 0. };
	double3 a_em = { 0., 0., 0. };

	GPU_PAIR_INTERACTION_WRAPPER(get_a(a_lj, a_em, p, _p););

	a_lj *= 48. * EPSILON * SIZE / SIGMA / SIGMA / M;
	a_em *= 1. / (4. * PI * EPSILON0) * Q * Q / SIZE / SIZE / M;

	double3 _a = acc.get(ind);
	double3 a = a_lj + a_em;
	acc.set(ind, a);

	vel.set(ind, v + TIME_STEP * a);
	pos.set(ind, p + TIME_STEP * (v + TIME_STEP * a));
}
__global__ void energy_gpu(vec pos, vec vel, double* energy) {
	double e_lj = 0;
	double e_em = 0;
	
	GPU_PAIR_INTERACTION_WRAPPER(get_e(e_lj, e_em, p, _p););

	e_lj *= 2. * EPSILON;
	e_em *= 1. / (8. * PI * EPSILON0) * Q * Q / SIZE;
	double e_k = M * hypot2(v) / 2.;
	energy[ind] = e_k + e_em + e_lj;
}

double get_energy() {
	static double total_energy = 0;
	if (energy_valid)
		return total_energy;
	energy_valid = true;

#ifndef __INTELLISENSE__
	energy_gpu <<< GRID_SIZE, BLOCK_SIZE >>> (pos, vel, energy);
#endif 

	static double _energy[AMOUNT];
	total_energy = 0;
	hipMemcpy(_energy, energy, AMOUNT * sizeof(double), hipMemcpyDeviceToHost);

	for (int i = 0; i < AMOUNT; i++) {
		total_energy += _energy[i];
	}

	return total_energy;
}
void euler_step() {

#ifndef __INTELLISENSE__
	euler_gpu << < GRID_SIZE, BLOCK_SIZE >> > (pos, vel, acc);
#endif

	pos_valid = vel_valid = energy_valid = false;
}
