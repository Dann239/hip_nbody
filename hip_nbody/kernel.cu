#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifndef __HIPCC__
#include ""
#include "hip/hip_runtime_api.h"
#endif

#ifdef __INTELLISENSE__
void __syncthreads() {}
#endif

#include "kernel.h"
#include "properties.h"

#include <iostream>
using namespace std;

#define POS(i) double3({posx[i], posy[i], posz[i]})
#define VEL(i) double3({velx[i], vely[i], velz[i]})

static double* pos[3];
static double* vel[3];
static double* energy;

void gpu_alloc() {
	for (int i = 0; i < 3; i++) {
		hipMalloc(&pos[i], AMOUNT * sizeof(double));
		hipMalloc(&vel[i], AMOUNT * sizeof(double));
	}
	hipMalloc(&energy, AMOUNT * sizeof(double));
}
void gpu_dealloc() {
	for (int i = 0; i < 3; i++) {
		hipFree(pos[i]);
		hipFree(vel[i]);
	}
	hipFree(energy);
	hipDeviceReset();
}

bool pos_valid = false, vel_valid = false, energy_valid = false;

void get_pos(double* _pos[3]) {
	if (pos_valid)
		return;
	for(int i = 0; i < 3; i++)
		hipMemcpy(_pos[i], pos[i], AMOUNT * sizeof(double), hipMemcpyDeviceToHost);
	pos_valid = true;
}
void get_vel(double* _vel[3]) {
	if (vel_valid)
		return;
	for (int i = 0; i < 3; i++)
		hipMemcpy(_vel[i], vel[i], AMOUNT * sizeof(double), hipMemcpyDeviceToHost);
	vel_valid = true;
}

void set_pos(double* _pos[3]) {
	for (int i = 0; i < 3; i++)
		hipMemcpy(pos[i], _pos[i], AMOUNT * sizeof(double), hipMemcpyHostToDevice);
	pos_valid = true;
}
void set_vel(double* _vel[3]) {
	for (int i = 0; i < 3; i++)
		hipMemcpy(vel[i], _vel[i], AMOUNT * sizeof(double), hipMemcpyHostToDevice);
	vel_valid = true;
}

void print_err() {
	hipDeviceSynchronize();
	cout << hipGetErrorString(hipGetLastError()) << endl;
}

__device__ double hypot2(double3 p) {
	return p.x * p.x + p.y * p.y + p.z * p.z;
}
__device__ double3 round(double3 a) {
	return { round(a.x),round(a.y),round(a.z) };
}
#ifndef __HIPCC__
__device__ double3& operator-= (double3& a, double3 b) {
	a.x -= b.x;
	a.y -= b.y;
	a.z -= b.z;
	return a;
}
__device__ double3& operator+=(double3& a, double3 b) {
	a.x += b.x;
	a.y += b.y;
	a.z += b.z;
	return a;
}
#endif
__device__ double3& operator*= (double3& a, double b) {
	a.x *= b;
	a.y *= b;
	a.z *= b;
	return a;
}
__device__ double3 operator- (double3 a, double3 b) {
	return { a.x - b.x, a.y - b.y, a.z - b.z };
}
__device__ double3 operator* (double b, double3 a) {
	return { a.x * b, a.y * b, a.z * b };
}
__device__ double3 operator+ (double3 a, double3 b) {
	return { a.x + b.x, a.y + b.y, a.z + b.z };
}


#define GPU_PAIR_INTERACTION_WRAPPER( __CODE__ ) 				\
																\
	int tid = threadIdx.x,										\
	bid = blockIdx.x,											\
	ind = bid * BLOCK_SIZE + tid;								\
																\
	double3 p = 1. / SIZE * POS(ind),							\
	v = VEL(ind);												\
																\
	__shared__ double3 _pos[BLOCK_SIZE];						\
	for (int i = 0; i < GRID_SIZE; i++) {						\
																\
		__syncthreads();										\
		_pos[tid] = 1. / SIZE * POS(i * BLOCK_SIZE + tid);		\
		__syncthreads();										\
																\
		for (int j = 0; j < BLOCK_SIZE; j++) {					\
			double3 _p = _pos[j];								\
			if (i != bid || j != tid) {							\
				__CODE__										\
			}													\
		}														\
	}															\

constexpr double ss_ss = (SIZE * SIZE) / (SIGMA * SIGMA);

__device__ void get_a(double3& a_lj, double3& a_em, double3 p, double3 _p) {
	double3 d = p - _p;
	d -= round(d);

	double d2 = hypot2(d),
		r2 = d2 * ss_ss,
		r_2 = 1. / r2,
		r_4 = r_2 * r_2,
		r_6 = r_4 * r_2,
		r_8 = r_4 * r_4,
		_2r_14__r_8 = (r_6 - .5) * r_8;
	
	a_lj += (_2r_14__r_8 * d);
	
	double d_1 = 1 / sqrt(d2);
	a_em += d_1 * d_1 * d_1 * d;
}
__device__ void get_e(double& e_lj, double& e_em, double3 p, double3 _p) {
	double3 d = p - _p;
	d -= round(d);

	double d2 = hypot2(d),
		r2 = d2 * ss_ss,
		r_2 = 1. / r2,
		r_4 = r_2 * r_2,
		r_6 = r_4 * r_2;

	e_lj += (r_6 - 1) * r_6;

	double d_1 = 1 / sqrt(d2);
	e_em += d_1;
}


__global__ void euler_gpu(double* posx, double* posy, double* posz, double* velx, double* vely, double* velz) {
	double3 a_lj = { 0., 0., 0. };
	double3 a_em = { 0., 0., 0. };
	
	GPU_PAIR_INTERACTION_WRAPPER(get_a(a_lj, a_em, p, _p););

	a_em = { 0., 0., 0. };

	v += (48. * EPSILON * SIZE * TIME_STEP / SIGMA / SIGMA / M) * a_lj + (1 / 4 / PI / EPSILON0 * Q * Q / SIZE / SIZE) * a_em;
	velx[ind] = v.x; vely[ind] = v.y, velz[ind] = v.z;
	v *= TIME_STEP;
	posx[ind] += v.x; posy[ind] += v.y, posz[ind] += v.z;
}
__global__ void energy_gpu(double* posx, double* posy, double* posz, double* velx, double* vely, double* velz, double* energy) {
	double e_lj = 0;
	double e_em = 0;
	double e_k = 0;
	GPU_PAIR_INTERACTION_WRAPPER(get_e(e_lj, e_em, p, _p););

	e_em = 0;

	e_lj *= 2. * EPSILON;
	e_em *= 1. / 8. / PI / EPSILON0 / SIZE * Q * Q;
	e_k += M * hypot2(v) / 2.;
	energy[ind] = e_k + e_em + e_lj;
}


double get_energy() {
	static double total_energy = 0;
	if (energy_valid)
		return total_energy;
	energy_valid = true;

#ifndef __INTELLISENSE__
	energy_gpu <<< GRID_SIZE, BLOCK_SIZE >>> (pos[X], pos[Y], pos[Z], vel[X], vel[Y], vel[Z], energy);
#endif 

	static double _energy[AMOUNT];
	total_energy = 0;
	hipMemcpy(_energy, energy, AMOUNT * sizeof(double), hipMemcpyDeviceToHost);

	for (int i = 0; i < AMOUNT; i++) {
		total_energy += _energy[i];
	}

	return total_energy;
}
void euler_step() {

#ifndef __INTELLISENSE__
	euler_gpu <<< GRID_SIZE, BLOCK_SIZE >>> (pos[X], pos[Y], pos[Z], vel[X], vel[Y], vel[Z]);
#endif

	pos_valid = vel_valid = energy_valid = false;
}
