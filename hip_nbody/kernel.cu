#include "hip/hip_runtime.h"

#ifndef __HIPCC__
#include ""
#endif

#include "butchers.cuh"
#include "kernel.h"

#include <iostream>
using namespace std;

static double* pos[3];
static double* vel[3];

void gpu_alloc() {
	for (int i = 0; i < 3; i++) {
		hipMalloc(&pos[i], AMOUNT * sizeof(double));
		hipMalloc(&vel[i], AMOUNT * sizeof(double));
	}
}
void gpu_dealloc() {
	for (int i = 0; i < 3; i++) {
		hipFree(pos[i]);
		hipFree(vel[i]);
	}
	hipDeviceReset();
}

void get_pos(double* _pos[3]) {
	for(int i = 0; i < 3; i++)
		hipMemcpy(_pos[i], pos[i], AMOUNT * sizeof(double), hipMemcpyDeviceToHost);
}
void get_vel(double* _vel[3]) {
	for (int i = 0; i < 3; i++)
		hipMemcpy(_vel[i], vel[i], AMOUNT * sizeof(double), hipMemcpyDeviceToHost);
}

void set_pos(double* _pos[3]) {
	for (int i = 0; i < 3; i++)
		hipMemcpy(pos[i], _pos[i], AMOUNT * sizeof(double), hipMemcpyHostToDevice);
}
void set_vel(double* _vel[3]) {
	for (int i = 0; i < 3; i++)
		hipMemcpy(vel[i], _vel[i], AMOUNT * sizeof(double), hipMemcpyHostToDevice);
}

void print_err() {
	hipDeviceSynchronize();
	cout << hipGetErrorString(hipGetLastError()) << endl;
}
