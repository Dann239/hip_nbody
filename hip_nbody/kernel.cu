#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#ifndef __HIPCC__
#include ""
#include "hip/hip_runtime_api.h"
#endif

#if defined __INTELLISENSE__
void __syncthreads() {}
#endif

#include "kernel.h"
#include "properties.h"

#include <iostream>
#include <stdio.h>
using namespace std;

#define d3_0 double3({0.,0.,0.})
hipStream_t stream = hipStreamDefault;

__host__ __device__ double3 extract(const double* const v[3], int i) {
	return double3({ v[X][i], v[Y][i], v[Z][i] });
}

class vec {
private:
	double* v_gpu_old[3];
	double* v_gpu_new[3];
	double* v_cpu[3];
	long long validity;
public:
	__device__ double3 get(int i) const {
		return extract(v_gpu_old, i);
	}
	__device__ void set(int i, double3 p) const {
		v_gpu_new[X][i] = p.x;
		v_gpu_new[Y][i] = p.y;
		v_gpu_new[Z][i] = p.z;
	}
	void gpu_copy() {
		for(int i = 0; i < 3; i++)
			hipMemcpyAsync(v_gpu_old[i], v_gpu_new[i], MEM_LEN, hipMemcpyDeviceToDevice, stream);
	}
	void invalidate() {
		for (int i = 0; i < 3; i++)
			hipMemcpyAsync(v_cpu[i], v_gpu_new[i], MEM_LEN, hipMemcpyDeviceToHost, stream);
		validity = false;
	}
	void init() {
		for (int i = 0; i < 3; i++) {
			hipMalloc(&v_gpu_old[i], MEM_LEN);
			hipMalloc(&v_gpu_new[i], MEM_LEN);
			hipHostMalloc(&v_cpu[i], MEM_LEN);
		}
		validity = true;
	}
	void get_all(double** v) {
		if (!validity) {
			hipStreamSynchronize(stream);
			for (int i = 0; i < 3; i++)
				swap(v[i], v_cpu[i]);
		}
		validity = true;
	}
	void set_all(double** v) {
		for (int i = 0; i < 3; i++)
			hipMemcpyAsync(v_gpu_old[i], v[i], MEM_LEN, hipMemcpyHostToDevice, stream);
		validity = true;
	}
	void destroy() {
		for (int i = 0; i < 3; i++) {
			hipHostFree(v_cpu[i]);
			hipFree(v_gpu_old[i]);
			hipFree(v_gpu_new[i]);
		}
	}
};

double* pos[3];
double* vel[3];

vec vec_pos, vec_vel;
static double* energy;
static double* _energy;
static properties* props;

double potential_energy = 0;
double kinetic_energy = 0;
double temperature = 0;
double total_energy = 0;

void alloc() {
	hipStreamCreate(&stream);

	for (int i = 0; i < 3; i++) {
		hipHostMalloc(&pos[i], MEM_LEN);
		hipHostMalloc(&vel[i], MEM_LEN);
	}

	vec_pos.init();
	vec_vel.init();

	hipMalloc(&energy, MEM_LEN);
	hipHostMalloc(&_energy, MEM_LEN);

	hipMalloc(&props, ELEMS_NUM * sizeof(properties));
	static properties* _props = (properties*)malloc(ELEMS_NUM * sizeof(properties));
	for(int i = 0; i < ELEMS_NUM; i++) _props[i].set_properties(ELEMS_TYPES[i]);
	hipMemcpy(props, _props, ELEMS_NUM * sizeof(properties), hipMemcpyHostToDevice);
}
void dealloc() {
	hipStreamDestroy(stream);
	for (int i = 0; i < 3; i++) {
		hipHostFree(pos[i]);
		hipHostFree(vel[i]);
	}
	vec_pos.destroy();
	vec_pos.destroy();
	hipFree(energy);
	hipFree(props);
	hipHostFree(_energy);
	hipDeviceReset();
}

void pull_values() {
	vec_pos.get_all(pos);
	vec_vel.get_all(vel);
}
void push_values() {
	vec_pos.set_all(pos);
	vec_vel.set_all(vel);
}

void print_err(bool force) {
	if(force)
		hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if(err || force)
		cout << hipGetErrorString(err) << endl;
}

__host__ __device__ bool invalid_elem(int block, properties p, int i) {
	return block / (double)GRID_SIZE < p.divisions[i];
}

__host__ __device__ int get_elem(int block, properties p) {
	for (int i = 1; i <= ELEMS_NUM; i++)
		if (invalid_elem(block, p, i))
			return i - 1;
	return ERROR;
}

properties get_properties(int num) {
	return properties(ELEMS_TYPES[get_elem(num / BLOCK_SIZE, properties(ERROR))]);
}

__host__ __device__ double hypot2(double3 p) {
	return p.x * p.x + p.y * p.y + p.z * p.z;
}
__device__ double3 round(double3 a) {
	return { round(a.x),round(a.y),round(a.z) };
}

#ifndef __HCC__
__device__ double3& operator-= (double3& a, double3 b) {
	a.x -= b.x;
	a.y -= b.y;
	a.z -= b.z;
	return a;
}
__device__ double3& operator+=(double3& a, double3 b) {
	a.x += b.x;
	a.y += b.y;
	a.z += b.z;
	return a;
}
#endif

__device__ double3& operator*= (double3& a, double b) {
	a.x *= b;
	a.y *= b;
	a.z *= b;
	return a;
}
__device__ double3 operator- (double3 a, double3 b) {
	return { a.x - b.x, a.y - b.y, a.z - b.z };
}
__device__ double3 operator* (double b, double3 a) {
	return { a.x * b, a.y * b, a.z * b };
}
__device__ double3 operator+ (double3 a, double3 b) {
	return { a.x + b.x, a.y + b.y, a.z + b.z };
}
__device__ double3 operator& (double3 a, double3 b) {
	return { a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x };
}

__device__ bool operator== (double3 a, double3 b) {
	return a.x == b.x && a.y == b.y && a.z == b.z;
}

__device__ void get_a(double3& a_lj, double3& a_em, double3 p, double3 _p, double ss_ss) {
	double3 d = p - _p;
	d -= round(d);

	double d2 = hypot2(d);

#ifdef ENABLE_LJ
	double r2 = d2 * ss_ss,
		r_2 = 1. / r2,
		r_4 = r_2 * r_2,
		r_6 = r_4 * r_2,
		r_8 = r_4 * r_4,
		_2r_14__r_8 = (r_6 - .5) * r_8;
	a_lj += (_2r_14__r_8 * d);
#endif

#ifdef ENABLE_EM
	double d_2 = 1. / d2,
		d_1 = sqrt(d_2);
	a_em += d_2 * d_1 * d;
#endif
}
__device__ void get_e(double& e_lj, double& e_em, double3 p, double3 _p, double ss_ss) {
	double3 d = p - _p;
	d -= round(d);

	double d2 = hypot2(d);

#ifdef ENABLE_LJ
	double r2 = d2 * ss_ss,
		r_2 = 1. / r2,
		r_4 = r_2 * r_2,
		r_6 = r_4 * r_2;
	e_lj += (r_6 - 1.) * r_6;
#endif

#ifdef ENABLE_EM
	double d_1 = 1. / sqrt(d2);
	e_em += d_1;
#endif
}

struct interaction_wrapper {
	properties _P0, _P;
	double epsilon, sigma;
	double lj_coeff[ELEMS_NUM];
	double em_coeff[ELEMS_NUM];
	double ss_ss[ELEMS_NUM];
	double3 p, _p;
	double3 v;
	int props_ind;
	double *_posx, *_posy, *_posz;

	template<typename T,typename T_, typename T__>
	__device__ interaction_wrapper(double3& p_, double3& v_, const vec vec_pos, const vec vec_vel, double _posx[], double _posy[], double _posz[], const properties* props, T coeffs, T_ body, T__ post) : _P0(props[0]), _P(props[0]) {

		int tid = threadIdx.x,
		bid = blockIdx.x,
		ind = bid * blockDim.x + tid;

		p = 1. / SIZE * vec_pos.get(ind);
		v = vec_vel.get(ind);

		_P0 = props[get_elem(bid, props[0])];

		for(props_ind = 0; props_ind < ELEMS_NUM; props_ind++) {
			_P = props[props_ind];
			epsilon = sqrt(_P.EPSILON * _P0.EPSILON);
			sigma = (_P.SIGMA + _P0.SIGMA) / 2;
			ss_ss[props_ind] = (SIZE * SIZE) / (sigma * sigma);
			coeffs(*this);
		}
		props_ind = 0;
		for (int i = 0; i < GRID_SIZE; i++) {

			__syncthreads();
			double3 _pos = 1. / SIZE * vec_pos.get(i * BLOCK_SIZE + tid);
			_posx[tid] = _pos.x; _posy[tid] = _pos.y; _posz[tid] = _pos.z;

			if ( invalid_elem(i, _P0, props_ind ))
				props_ind++;

			__syncthreads();
			for (int j = 0; j < BLOCK_SIZE; j++) {
				_p = double3({_posx[j],_posy[j],_posz[j]});
				if (i != bid || j != tid)
					body(*this);

			}
			post(*this);
		}

		p *= SIZE;

		p_ = p;
		v_ = v;
	}
};

__global__
void euler_gpu(const vec vec_pos, const vec vec_vel, const properties* props) {
	extern __shared__ double _posx[];
	extern __shared__ double _posy[];
	extern __shared__ double _posz[];
	
	int tid = threadIdx.x,
		bid = blockIdx.x,
		ind = bid * blockDim.x + tid;

	double3 a_lj = d3_0, a_em = d3_0, da_lj = d3_0, da_em = d3_0;
	double3 p, v;

	interaction_wrapper(p, v, vec_pos, vec_vel, _posx, _posy, _posz, props,
		[&](interaction_wrapper& w) {
			w.lj_coeff[w.props_ind] = 48. * w.epsilon * SIZE / w.sigma / w.sigma / w._P0.M;
			w.em_coeff[w.props_ind] = 1. / (4. * PI * EPSILON0) * w._P0.Q * w._P.Q / SIZE / SIZE / w._P0.M;
		},
		[&](interaction_wrapper& w) {
			get_a(da_lj, da_em, w.p, w._p, w.ss_ss[w.props_ind]);
		},
		[&](interaction_wrapper& w) {
			a_lj += w.lj_coeff[w.props_ind] * da_lj;
			a_em += w.em_coeff[w.props_ind] * da_em;
			da_lj = da_em = d3_0;
		}
	);

	double3 a = a_lj + a_em;

	v += TIME_STEP * a;
	p += TIME_STEP * v;

	vec_pos.set(ind, p);
	vec_vel.set(ind, v);

}

__global__
void energy_gpu (const vec vec_pos, const vec vec_vel, double* energy, const properties* props) {
	extern __shared__ double _posx[];
	extern __shared__ double _posy[];
	extern __shared__ double _posz[];

	int tid = threadIdx.x,
		bid = blockIdx.x,
		ind = bid * blockDim.x + tid;

	double e_lj = 0, e_em = 0, de_lj = 0, de_em = 0;
	double3 p, v;

	interaction_wrapper(p, v, vec_pos, vec_vel, _posx, _posy, _posz, props,
		[&](interaction_wrapper& w) {
			w.lj_coeff[w.props_ind] = 2. * w.epsilon;
			w.em_coeff[w.props_ind] = 1. / (8. * PI * EPSILON0) * w._P0.Q * w._P.Q / SIZE;
		},
		[&](interaction_wrapper& w) {
			get_e(de_lj, de_em, w.p, w._p, w.ss_ss[w.props_ind]);
		},
		[&](interaction_wrapper& w) {
			e_lj += w.lj_coeff[w.props_ind] * de_lj;
			e_em += w.em_coeff[w.props_ind] * de_em;
			de_lj = de_em = 0;
		}
	);

	energy[ind] = e_em + e_lj;
}

void energy_calc() {
	potential_energy = 0;
	kinetic_energy = 0;
	for (int i = 0; i < AMOUNT; i++) {
		potential_energy += _energy[i] / AMOUNT;
		kinetic_energy += get_properties(i).M * hypot2(extract(vel, i)) / 2. / AMOUNT;
	}
	total_energy = potential_energy + kinetic_energy;
}

void euler_steps(int steps) {
	for(int i = 0; i < steps; i++) {
	#ifndef __INTELLISENSE__
		euler_gpu <<< GRID_SIZE, BLOCK_SIZE, sizeof(double) * BLOCK_SIZE * 3, stream >>> (vec_pos, vec_vel, props);
	#endif
		vec_pos.gpu_copy();
		vec_vel.gpu_copy();
	}
	vec_pos.invalidate();
	vec_vel.invalidate();

	energy_calc();

#ifndef __INTELLISENSE__
	energy_gpu <<< GRID_SIZE, BLOCK_SIZE, sizeof(double) * BLOCK_SIZE * 3, stream >>> (vec_pos, vec_vel, energy, props);
#endif
	hipMemcpyAsync(_energy, energy, MEM_LEN, hipMemcpyDeviceToHost, stream);

}
void force_energy_calc() {
#ifndef __INTELLISENSE__
	energy_gpu <<< GRID_SIZE, BLOCK_SIZE, sizeof(double) * BLOCK_SIZE * 3 >>> (vec_pos, vec_vel, energy, props);
#endif
	hipMemcpy(_energy, energy, MEM_LEN, hipMemcpyDeviceToHost);
	energy_calc();
}

void print_chars() {
	hipDeviceProp_t chars;

	hipGetDeviceProperties(&chars, 0);
	printf("Device:\n");
	printf("major: %d\n", chars.major);
	printf("minor: %d\n", chars.minor);
	printf("canMapHostMemory: %d\n", chars.canMapHostMemory);
	printf("multiProcessorCount: %d\n", chars.multiProcessorCount);
	printf("sharedMemPerBlock: %zu\n", chars.sharedMemPerBlock);
	printf("maxThreadsDim: %d\n", chars.maxThreadsDim[0]);
	printf("maxThreadsPerMultiProcessor: %d\n", chars.maxThreadsPerMultiProcessor);
	printf("regsPerBlock: %d\n\n", chars.regsPerBlock);

#ifndef __HIPCC__
	printf("singleToDoublePrecisionPerfRatio: %d\n", chars.singleToDoublePrecisionPerfRatio);
	printf("kernelExecTimeoutEnabled: %d\n", chars.kernelExecTimeoutEnabled);
	printf("regsPerMultiprocessor: %d\n", chars.regsPerMultiprocessor);
	printf("sharedMemPerMultiprocessor: %zu\n", chars.sharedMemPerMultiprocessor);
	printf("warpSize: %d\n\n", chars.warpSize);
#endif
	hipFuncAttributes attr;
	hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(euler_gpu));
	printf("euler_gpu:\n");
	printf("binaryVersion: %d\n", attr.binaryVersion);
	printf("ptxVersion: %d\n", attr.ptxVersion);
	printf("maxThreadsPerBlock: %d\n", attr.maxThreadsPerBlock);
	printf("numRegs: %d\n", attr.numRegs);
	printf("localSizeBytes: %zu\n", attr.localSizeBytes);
	printf("sharedSizeBytes: %zu\n", attr.sharedSizeBytes);

#ifndef __HCC__
	int numBlocks;
	hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, (const void*)euler_gpu, BLOCK_SIZE, sizeof(double) * BLOCK_SIZE * 3);
	printf("BlockSize = %d; BlocksPerMP = %d; Occupancy = %f\n", BLOCK_SIZE, numBlocks, (double) (numBlocks * BLOCK_SIZE) / (chars.maxThreadsPerMultiProcessor));

	printf("\n");
#endif
}
