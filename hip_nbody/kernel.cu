#include "hip/hip_runtime.h"

#ifndef __HIPCC__
#include ""
#endif

#include "butchers.cuh"
#include "kernel.h"

#include <iostream>
using namespace std;

static double* p[3];
static double* v[3];

void gpu_alloc() {
	for (int i = 0; i < 3; i++) {
		hipMalloc(&p[i], AMOUNT * sizeof(double));
		hipMalloc(&v[i], AMOUNT * sizeof(double));
	}
}
void gpu_dealloc() {
	for (int i = 0; i < 3; i++) {
		hipFree(p[i]);
		hipFree(v[i]);
	}
	hipDeviceReset();
}

void get_pos(double* _p[3]) {
	for(int i = 0; i < 3; i++)
		hipMemcpy(_p[i], p[i], AMOUNT * sizeof(double), hipMemcpyDeviceToHost);
}
void get_vel(double* _v[3]) {
	for (int i = 0; i < 3; i++)
		hipMemcpy(_v[i], v[i], AMOUNT * sizeof(double), hipMemcpyDeviceToHost);
}

void set_pos(double* _p[3]) {
	for (int i = 0; i < 3; i++)
		hipMemcpy(p[i], _p[i], AMOUNT * sizeof(double), hipMemcpyHostToDevice);
}
void set_vel(double* _v[3]) {
	for (int i = 0; i < 3; i++)
		hipMemcpy(v[i], _v[i], AMOUNT * sizeof(double), hipMemcpyHostToDevice);
}

void print_err() {
	hipDeviceSynchronize();
	cout << hipGetErrorString(hipGetLastError()) << endl;
}
